#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

#include <hipcub/hipcub.hpp>

template<typename T, typename O>
__global__ void complete_cumsum(T *input, O *output, int64_t *total) {
  int64_t const v = *output + *input;
  *total = *(output + 1) = v;
}



template <typename T, typename O>
int64_t full_cumsum_helper(T *input, int64_t input_size, O *output) {
  size_t temp_storage_bytes = 0;
  auto stream = at::cuda::getCurrentCUDAStream();


  hipcub::DeviceScan::ExclusiveSum(nullptr, temp_storage_bytes, input, output, input_size, stream);

  // Make temp storage on in the torch mempool.
  auto temp_storage = torch::empty({int64_t(temp_storage_bytes)}, 
    torch::TensorOptions().dtype(torch::kUInt8).device(torch::kCUDA));

  hipcub::DeviceScan::ExclusiveSum(temp_storage.data_ptr<uint8_t>(), 
    temp_storage_bytes, input, output, input_size, stream);
    
  // cumsum is now in the output but the final value is missing.

  // Make storage in pinned_memory.
  auto total_tensor_options = torch::TensorOptions().dtype(torch::kInt64).pinned_memory(true);
  auto total_tensor = torch::empty({int64_t(1)}, total_tensor_options);

  // Now we run a kernel that will complete the full_cumsum.  The
  // final element of input gets added to the penultimate element of
  // output and written into the final element of output.  It also
  // gets written into total_tensor.
  complete_cumsum<<<1,1>>>(&input[input_size-1],
                           &output[input_size-1],
                           total_tensor.data_ptr<int64_t>());
  // complete_cumsum must finish before we can read the total_tensor.
  hipDeviceSynchronize();
  return *total_tensor.data_ptr<int64_t>();
}


// full cumsum and also returns the total to the CPU.
int64_t full_cumsum(const torch::Tensor input, const torch::Tensor output) {
  assert(input.size(0) + 1 == output.size(0));
  switch (input.scalar_type()) {
    case torch::kInt16:
      return full_cumsum_helper(input.data_ptr<int16_t>(), input.size(0),
                                output.data_ptr<int32_t>());
      break;

    case torch::kInt32:
      return full_cumsum_helper(input.data_ptr<int32_t>(), input.size(0),
                                output.data_ptr<int32_t>());
      break;
    case torch::kInt64:
      return full_cumsum_helper(input.data_ptr<int64_t>(), input.size(0),
                                output.data_ptr<int64_t>());
      break;
    default:
      // TODO, add all the other cases.
      throw std::runtime_error("Not yet implemented for data type.");
      break;
  }
}




