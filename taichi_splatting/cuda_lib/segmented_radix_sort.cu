#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

#include <hipcub/hipcub.hpp>


// for our workload segmented sort (non radix) seems faster
template <typename K, typename V>
void sort_helper(
  K *d_keys, V *d_values, 
  K *d_keys_out, V *d_values_out,

  int num_items, 
  int64_t *d_start_offset, int64_t *d_end_offset, 
  int num_segments,   int begin_bit=0, int end_bit=-1)  
{

  size_t   temp_storage_bytes = 0;
  end_bit = end_bit > 0 ? end_bit : sizeof(K) * 8;
  auto stream = at::cuda::getCurrentCUDAStream();

  hipcub::DeviceSegmentedRadixSort::SortPairs(nullptr, temp_storage_bytes,
      d_keys, d_keys_out, d_values, d_values_out,
      num_items, num_segments, 
      d_start_offset, d_end_offset, 
      begin_bit, end_bit, stream);

  auto temp_storage = torch::empty({int64_t(temp_storage_bytes)}, 
    torch::TensorOptions().dtype(torch::kUInt8).device(torch::kCUDA));

  hipcub::DeviceSegmentedRadixSort::SortPairs(temp_storage.data_ptr<uint8_t>(), temp_storage_bytes,
      d_keys, d_keys_out, d_values, d_values_out,
      num_items, num_segments, 
      d_start_offset, d_end_offset, 
      begin_bit, end_bit, stream);

}


std::pair<torch::Tensor, torch::Tensor> segmented_radix_sort(
  const torch::Tensor keys, const torch::Tensor values,
  const torch::Tensor start_offset, const torch::Tensor end_offset,
  int begin_bit=0, int end_bit=-1) {
  
  assert (keys.dim() == 1 && values.dim() == 1), "keys and values must be 1D";
  assert (keys.size(0) == values.size(0)), "keys and values must have the same size";

  assert (start_offset.dim() == 1 && end_offset.dim() == 1 && start_offset.size(0) == end_offset.size(0)), 
    "start_offset and end_offset must be 1D and have the same size"; 

  assert (start_offset.scalar_type() == torch::kInt64 
    && end_offset.scalar_type() == torch::kInt64), "start_offset/end_offset must be int64";
  
  auto keys_out = torch::empty_like(keys);
  auto values_out = torch::empty_like(values);

  if (keys.scalar_type() == torch::kInt32 && values.scalar_type() == torch::kInt32) {
    sort_helper<int32_t, int32_t>(
      keys.data_ptr<int32_t>(), values.data_ptr<int32_t>(), 
      keys_out.data_ptr<int32_t>(), values_out.data_ptr<int32_t>(),
      keys.size(0), 
      start_offset.data_ptr<int64_t>(), end_offset.data_ptr<int64_t>(),
      start_offset.size(0), begin_bit, end_bit);

      return std::make_pair(keys_out, values_out);
  } else if (keys.scalar_type() == torch::kInt16 && values.scalar_type() == torch::kInt32) {
    sort_helper<int16_t, int32_t>(
      keys.data_ptr<int16_t>(), values.data_ptr<int32_t>(), 
      keys_out.data_ptr<int16_t>(), values_out.data_ptr<int32_t>(),
      keys.size(0), 
      start_offset.data_ptr<int64_t>(), end_offset.data_ptr<int64_t>(),
      start_offset.size(0), begin_bit, end_bit);

      return std::make_pair(keys_out, values_out);
  } else { 
      // TODO, add all the other cases.
      throw std::runtime_error("Not yet implemented for data type.");
  }
}

